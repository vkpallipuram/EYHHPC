#include "hip/hip_runtime.h"
/**************************************************************************************
 *The gpu.cu performs are the cannyEdge detection,Vertical and Horizontal Edge keeping
 *and mean keeping to obtain a High resolution image from low resolution image input
 *Various kernel functions are invoked to perform the above mentioned operations.
**************************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"hip/hip_runtime_api.h"
#include<math.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#include "gpu.h"


__constant__ float *d_gx_mask;
__constant__ float *d_gy_mask;
__constant__ float *mask;


__shared__ float Nshared[BLOCKWIDTH+2*(g_w/2)][BLOCKWIDTH+2*(g_w/2)];
__device__
void prepNshared(float *image,int width,int height){

                int i,j;
                //NORTH
                if(blockIdx.x==0)
                {
                        if(threadIdx.x >= BLOCKWIDTH - g_w/2)
                        {
                                Nshared[threadIdx.x+g_w/2 - BLOCKWIDTH][threadIdx.y+g_w/2]=0;
                        }
                }
                else
                {
                        if(threadIdx.x >=BLOCKWIDTH - g_w/2)
                        {
                                i=threadIdx.x+(blockIdx.x-1)*blockDim.x;
                                j=threadIdx.y+blockIdx.y*blockDim.y;
                                Nshared[threadIdx.x+g_w/2 - BLOCKWIDTH][threadIdx.y+g_w/2]=
                                image[i*width+j];
                        }
                }

		//for south elements

                if(blockIdx.x==gridDim.x - 1)
                {
                        if(threadIdx.x < g_w/2)
                        {
                                 Nshared[threadIdx.x + BLOCKWIDTH + g_w/2][threadIdx.y+g_w/2]=0; 
                        }
                }
                else
                {
                        if(threadIdx.x < g_w/2)
                        {
                                i= threadIdx.x +(blockIdx.x+1)*blockDim.x;
                                j=threadIdx.y+(blockIdx.y *blockDim.y);
                                Nshared[threadIdx.x + BLOCKWIDTH + g_w/2][threadIdx.y+g_w/2]=image[i*width+j];
                        }
                }
                //for west elements

                if(blockIdx.y==0)
                {
                        if(threadIdx.y>= BLOCKWIDTH-g_w/2)
                        {
                                Nshared[threadIdx.x +g_w/2][threadIdx.y+g_w/2 - BLOCKWIDTH]=0;
                        }
                }
                else
                {
                        if(threadIdx.y>= BLOCKWIDTH-g_w/2)
                        {
                                i= threadIdx.x +blockIdx.x *blockDim.x;
                                j = threadIdx.y+(blockIdx.y-1)*blockDim.y;
                                Nshared[threadIdx.x +g_w/2][threadIdx.y+g_w/2 - BLOCKWIDTH]=image[i*width+j];
                        }
                }


                //for east elements

                if(blockIdx.y==gridDim.y-1)
                {
                        if(threadIdx.y < g_w/2) //VKP:Initially BLOCKWIDTH-g_w/2
                        {
                                Nshared[threadIdx.x +g_w/2][threadIdx.y+g_w/2+BLOCKWIDTH]=0;
                        }
                }
                else
                {
                        if(threadIdx.y < g_w/2) //VKP:Initially BLOCKWIDTH-g_w/2
                        {
                                i=threadIdx.x+blockIdx.x*blockDim.x;
                                j = threadIdx.y +(blockIdx.y+1)*blockDim.y;
                                Nshared[threadIdx.x +g_w/2][threadIdx.y+g_w/2+BLOCKWIDTH]=image[i*width+j];
                        }
                }

                //for north west elements

                if(blockIdx.x==0 || blockIdx.y==0)
                {
                        if((threadIdx.x >= BLOCKWIDTH-g_w/2)&&(threadIdx.y >= BLOCKWIDTH - g_w/2))
                        {
                                Nshared[threadIdx.x+g_w/2-BLOCKWIDTH][threadIdx.y+g_w/2 - BLOCKWIDTH]=0;
                        }
                }
                else
                {
                        if((threadIdx.x >= BLOCKWIDTH-g_w/2)&&(threadIdx.y >= BLOCKWIDTH - g_w/2))
                        {
                                i= threadIdx.x+(blockIdx.x -1)*blockDim.x;
                                j = threadIdx.y +(blockIdx.y -1)*blockDim.y;
                                Nshared[threadIdx.x+g_w/2-BLOCKWIDTH][threadIdx.y+g_w/2 - BLOCKWIDTH]=image[i*width+j];
                        }
                }
                //for north east elements

                if((blockIdx.x==0) || (blockIdx.y == gridDim.y-1))
                {
                        if((threadIdx.x >= BLOCKWIDTH - g_w/2)&&(threadIdx.y <g_w/2))
                        {
                                Nshared[threadIdx.x +g_w/2-BLOCKWIDTH][threadIdx.y+g_w/2+BLOCKWIDTH]=0; 
                        }
                }
                else
                {
                        if((threadIdx.x >= BLOCKWIDTH - g_w/2)&&(threadIdx.y <g_w/2))
                        {
                                i= threadIdx.x +(blockIdx.x -1)*blockDim.x;
                                j = threadIdx.y +(blockIdx.y +1)*blockDim.y;
                                Nshared[threadIdx.x +g_w/2 - BLOCKWIDTH][threadIdx.y+g_w/2+BLOCKWIDTH]=image[i*width+j] =image[i*width+j]; 
                        }
                }

                //for south west elements

                if((blockIdx.x==gridDim.x-1)||(blockIdx.y==0))
                {
                        if((threadIdx.x<g_w/2)&&(threadIdx.y >=BLOCKWIDTH- g_w/2)) 
                        {
                                Nshared[threadIdx.x+g_w/2+BLOCKWIDTH][threadIdx.y+g_w/2 - BLOCKWIDTH] =0; 
                        }
                }
                else
                {
                        if((threadIdx.x <g_w/2)&&(threadIdx.y>= BLOCKWIDTH-g_w/2))
                        {
                                i= threadIdx.x +(blockIdx.x +1)*blockDim.x;
                                j = threadIdx.y +(blockIdx.y -1)*blockDim.y;
                                Nshared[threadIdx.x+g_w/2+BLOCKWIDTH][threadIdx.y+g_w/2 - BLOCKWIDTH] =image[i*width+j];
                        }
                }

                //forsouth east elements

                if((blockIdx.x == gridDim.x -1) || (blockIdx.y == gridDim.y-1))
                {
                        if((threadIdx.x <g_w/2)&&(threadIdx.y <g_w/2))
                        {
                                Nshared[threadIdx.x +g_w/2+BLOCKWIDTH][threadIdx.y+g_w/2+BLOCKWIDTH]=0;
                        }
                }
                else
                {
                        if((threadIdx.x <g_w/2)&&(threadIdx.y <g_w/2))
                        {
                                i=threadIdx.x +(blockIdx.x+1)*blockDim.x;
                                j = threadIdx.y+(blockIdx.y+1)*blockDim.y;
                        Nshared[threadIdx.x +g_w/2+BLOCKWIDTH][threadIdx.y+g_w/2+BLOCKWIDTH]=image[i*width+j];
                        }
                }

                i= threadIdx.x+blockIdx.x*blockDim.x;
                j=threadIdx.y+blockIdx.y*blockDim.y;

                Nshared[threadIdx.x +(g_w/2)][threadIdx.y+(g_w/2)]=image[i*width+j];

                __syncthreads ();

}

__device__
int gpu_compare(int v,int d, int ad)
{
	int a=0; //default vertical

	if(v>d)
	{
		if(v>ad)
			a=0;
		else
			a=-1;

	}
	else
	{
		if(d>ad)
			a=1;
		else
			a=-1;
	}
	return a;
}


//kernel for magAngle and suppression
__global__
void gpu_conv_mag_phase(float *image,int width,int height,float *d_Gxy_outimage,float *d_Igx_mask,float *d_Igy_mask,float *d_I_angle,float *d_suppressed){
        int i,j;
        float temp=0;
        float temp1=0;
        int tid1 = threadIdx.x+blockIdx.x*blockDim.x;
        int tid2= threadIdx.y+blockIdx.y*blockDim.y;
        if((threadIdx.x+blockIdx.x*blockDim.x)<height && (threadIdx.y + blockIdx.y*blockDim.y)<width)
        {
          prepNshared(image,width,height);

         for(i=0;i<g_w;i++)
          {
                for(j=0;j<g_w;j++)
                {
                        temp = temp+d_Igx_mask[i*g_w+j]*Nshared[i+threadIdx.x][j+threadIdx.y];
                        temp1 = temp1+d_Igy_mask[i*g_w+j]*Nshared[i+threadIdx.x][j+threadIdx.y];
                }
        }
        d_Gxy_outimage[tid1*width+tid2]=sqrt(temp*temp+temp1*temp1);
        d_suppressed[tid1*width+tid2]=sqrt(temp*temp+temp1*temp1);
        d_I_angle[tid1*width+tid2] = atan2(temp, temp1);
 	} 
}


__global__
void gpu_doubleThreshold(float *d_hyst,float *d_suppressed,int width,int height,float th_high,float th_low){

int tid1 = threadIdx.x+blockIdx.x*blockDim.x;
int tid2 = threadIdx.y+blockIdx.y*blockDim.y;

	if(tid1<height && tid2<width)
	{
		if (d_suppressed[tid1*width+tid2]>th_high)
		{
			d_hyst[tid1*width+tid2]=(float)255;
			d_suppressed[tid1*width+tid2]=(float)255;
		}
		else if (d_suppressed[tid1*width+tid2]<th_high && d_suppressed[tid1*width+tid2]>th_low)
		{
			d_hyst[tid1*width+tid2]=(float)125;
			d_suppressed[tid1*width+tid2]=(float)125;
		}
		else 
		{
			d_hyst[tid1*width+tid2]=0;
			d_suppressed[tid1*width+tid2]=0;
		}
	}
}

//edge linking

__global__
void gpu_edgeLinking(float *d_buffer,float *d_hyst,int width,int height){

int tid1 = threadIdx.x+blockIdx.x*blockDim.x;
int tid2 = threadIdx.y+blockIdx.y*blockDim.y;

        if(tid1<height && tid2<width)
	{	
		if(d_buffer[tid1*width+tid2]==(float)125)
		{	
			d_hyst[tid1*width+tid2]=(float)0;
			if(tid1-1>=0)
			{
                           	if(d_buffer[(tid1-1)*width +tid2]==(float)255)
				{
                                      d_hyst[tid1*width+tid2]=(float)255;
				}
			}
                        if(tid1+1<height)
			{
                              	if(d_buffer[(tid1+1)*width + tid2]==(float)255)
			      	{
                                   d_hyst[tid1*width+tid2]=(float)255;
				}	
			}
				
			//left and right (i,j-1) and (i,j+1)
			if(tid1-1>=0)
			{
                        	if(d_buffer[tid1*width + (tid2-1)]==(float)255)
				{
                                	d_hyst[tid1*width + tid2]=(float)255;
				}
			}
                        if(tid2+1<width)
			{
                        	if(d_buffer[tid1*width + (tid2+1)]==255)
				{
                                	d_hyst[tid1*width + tid2]=(float)255;
				}
			}
				
			//Diagonal Pixels (tid1-1,tid2-1) and (tid1+1,tid2+1)
			if((tid1-1) >=0 && (tid2-1) >=0)
			{
                        	if(d_buffer[tid1*width + (tid2-1)]==(float)255)
				{
                                	d_hyst[tid1*width + tid2]=(float)255;
				}
			}
                        if(tid1+1<height && (tid2+1) <width)
			{
                        	if(d_buffer[(tid1+1)*width + (tid2+1)]==(float)255)
				{
                                	d_hyst[tid1*width + tid2]=(float)255;
				}
			}
			
			//Anti-Diagonal Pixels (tid1-1,tid2+1) (tid1+1,tid2-1)
			if((tid1-1)>=0 && (tid2+1) <width)
			{
                        	if(d_buffer[(tid1-1)*width + (tid2+1)]==(float)255)
				{
                                	d_hyst[tid1*width + tid2]=(float)255;
				}
			}
                        if((tid1+1)<height && (tid2-1) >=0)
			{
                        	if(d_buffer[(tid1+1)*width + (tid2-1)]==(float)255)
				{
                                	d_hyst[tid1*width + tid2]=(float)255;
				}
			}
                  }
	}
}


//device shared kernel for nonmaximal suppression
__shared__ float Nshared1[BLOCKWIDTH+2*(apron/2)][BLOCKWIDTH+2*(apron/2)];
__device__
void prepNshared1(float *image,int width,int height){

                int i,j;
                //NORTH
                if(blockIdx.x==0)
                {
                        if(threadIdx.x >= BLOCKWIDTH - apron/2)
                        {
                                Nshared1[threadIdx.x+apron/2 - BLOCKWIDTH][threadIdx.y+apron/2]=0;
                        }
                }
                else
                {
                        if(threadIdx.x >=BLOCKWIDTH - apron/2)
                        {
                                i=threadIdx.x+(blockIdx.x-1)*blockDim.x;
                                j=threadIdx.y+blockIdx.y*blockDim.y;
                                Nshared1[threadIdx.x+apron/2 - BLOCKWIDTH][threadIdx.y+apron/2]=
                                image[i*width+j];
                        }
                }

                //for south elements

                if(blockIdx.x==gridDim.x - 1)
                {
                        if(threadIdx.x < apron/2)
                        {
                                 Nshared1[threadIdx.x + BLOCKWIDTH + apron/2][threadIdx.y+apron/2]=0; 
                        }
                }
                else
                {
                        if(threadIdx.x < apron/2)
                        {
                                i= threadIdx.x +(blockIdx.x+1)*blockDim.x;
                                j=threadIdx.y+(blockIdx.y *blockDim.y);
                                Nshared1[threadIdx.x + BLOCKWIDTH + apron/2][threadIdx.y+apron/2]=image[i*width+j];//VKP:Initially theadIdx.x + BLOCKWIDTH - apron/2
                        }
                }
                //for west elements
if(blockIdx.y==0)
                {
                        if(threadIdx.y>= BLOCKWIDTH-apron/2)
                        {
                                Nshared1[threadIdx.x +apron/2][threadIdx.y+apron/2 - BLOCKWIDTH]=0;
                        }
                }
                else
                {
                        if(threadIdx.y>= BLOCKWIDTH-apron/2)
                        {
                                i= threadIdx.x +blockIdx.x *blockDim.x;
                                j = threadIdx.y+(blockIdx.y-1)*blockDim.y;
                                Nshared1[threadIdx.x +apron/2][threadIdx.y+apron/2 - BLOCKWIDTH]=image[i*width+j];
                        }
                }


                //for east elements

                if(blockIdx.y==gridDim.y-1)
                {
                        if(threadIdx.y < apron/2) //VKP:Initially BLOCKWIDTH-apron/2
                        {
                                Nshared1[threadIdx.x +apron/2][threadIdx.y+apron/2+BLOCKWIDTH]=0;
                        }
                }
                else
                {
                        if(threadIdx.y < apron/2) //VKP:Initially BLOCKWIDTH-apron/2
                        {
                                i=threadIdx.x+blockIdx.x*blockDim.x;
                                j = threadIdx.y +(blockIdx.y+1)*blockDim.y;
                                Nshared1[threadIdx.x +apron/2][threadIdx.y+apron/2+BLOCKWIDTH]=image[i*width+j];
                        }
                }

                //for north west elements

                if(blockIdx.x==0 || blockIdx.y==0)
                {
                        if((threadIdx.x >= BLOCKWIDTH-apron/2)&&(threadIdx.y >= BLOCKWIDTH - apron/2))
                        {
                                Nshared1[threadIdx.x+apron/2-BLOCKWIDTH][threadIdx.y+apron/2 - BLOCKWIDTH]=0;
                        }
                }
                else
                {
                        if((threadIdx.x >= BLOCKWIDTH-apron/2)&&(threadIdx.y >= BLOCKWIDTH - apron/2))
                        {
                                i= threadIdx.x+(blockIdx.x -1)*blockDim.x;
                                j = threadIdx.y +(blockIdx.y -1)*blockDim.y;
                                Nshared1[threadIdx.x+apron/2-BLOCKWIDTH][threadIdx.y+apron/2 - BLOCKWIDTH]=image[i*width+j];
                        }
                }
//for north east elements

                if((blockIdx.x==0) || (blockIdx.y == gridDim.y-1))
                {
                        if((threadIdx.x >= BLOCKWIDTH - apron/2)&&(threadIdx.y <apron/2))
                        {
                                Nshared1[threadIdx.x +apron/2-BLOCKWIDTH][threadIdx.y+apron/2+BLOCKWIDTH]=0; 
                        }
                }
                else
                {
                        if((threadIdx.x >= BLOCKWIDTH - apron/2)&&(threadIdx.y <apron/2))
                        {
                                i= threadIdx.x +(blockIdx.x -1)*blockDim.x;
                                j = threadIdx.y +(blockIdx.y +1)*blockDim.y;
                                Nshared1[threadIdx.x +apron/2 - BLOCKWIDTH][threadIdx.y+apron/2+BLOCKWIDTH]=image[i*width+j] =image[i*width+j]; 
                        }
                }

                //for south west elements

                if((blockIdx.x==gridDim.x-1)||(blockIdx.y==0))
                {
                        if((threadIdx.x<apron/2)&&(threadIdx.y >=BLOCKWIDTH- apron/2)) 
                        {
                                Nshared1[threadIdx.x+apron/2+BLOCKWIDTH][threadIdx.y+apron/2 - BLOCKWIDTH] =0; 
                        }
                }
                else
                {
                        if((threadIdx.x <apron/2)&&(threadIdx.y>= BLOCKWIDTH-apron/2))
                        {
                                i= threadIdx.x +(blockIdx.x +1)*blockDim.x;
                                j = threadIdx.y +(blockIdx.y -1)*blockDim.y;
                                Nshared1[threadIdx.x+apron/2+BLOCKWIDTH][threadIdx.y+apron/2 - BLOCKWIDTH] =image[i*width+j]; 
                        }
                }

                //forsouth east elements

                if((blockIdx.x == gridDim.x -1) || (blockIdx.y == gridDim.y-1))
                {
                        if((threadIdx.x <apron/2)&&(threadIdx.y <apron/2))
                        {
                                Nshared1[threadIdx.x +apron/2+BLOCKWIDTH][threadIdx.y+apron/2+BLOCKWIDTH]=0;
                        }
                }
                else
                {
                        if((threadIdx.x <apron/2)&&(threadIdx.y <apron/2))
                        {
                                i=threadIdx.x +(blockIdx.x+1)*blockDim.x;
                                j = threadIdx.y+(blockIdx.y+1)*blockDim.y;
                        Nshared1[threadIdx.x +apron/2+BLOCKWIDTH][threadIdx.y+apron/2+BLOCKWIDTH]=image[i*width+j];
                        }
                }

                i= threadIdx.x+blockIdx.x*blockDim.x;
                j=threadIdx.y+blockIdx.y*blockDim.y;

                Nshared1[threadIdx.x +(apron/2)][threadIdx.y+(apron/2)]=image[i*width+j];
 		__syncthreads ();

}             

//non maximal suppression

__global__
void gpu_suppress(float *d_Gxy_outimage,float *d_suppressed,float *d_I_angle,int width,int height){
	float theta=0;
        float mag=0;
        int tid1 = threadIdx.x+blockIdx.x*blockDim.x;
        int tid2= threadIdx.y+blockIdx.y*blockDim.y;
        if((threadIdx.x+blockIdx.x*blockDim.x)<height && (threadIdx.y + blockIdx.y*blockDim.y)<width)
        {
          prepNshared1(d_Gxy_outimage,width,height);

         theta=(180/M_PI)*d_I_angle[tid1*width+tid2];
                        if(theta<0)
                                theta+=(float)180;
                        mag=Nshared1[threadIdx.x+(apron/2)][threadIdx.y+(apron/2)];
                        if(theta>(157.5) || theta <=22.5) //Left and Right
                        {
                                if((tid2-1) >=0)
				{
                                  if(mag<Nshared1[threadIdx.x+(apron/2)][threadIdx.y+(apron/2) -1])
                        	  {         
			             d_suppressed[tid1*width+tid2]=(float)0;
				  }
				}
                                if((tid2+1)<width)
				{
                                        if(mag<Nshared1[threadIdx.x+(apron/2)][threadIdx.y+(apron/2)+1])
					{
                                                d_suppressed[tid1*width+tid2]=(float)0;
                        		}
				}

			}

                        if(theta>(22.5) && theta <=67.5) //Diagonal pixels
                        {
                                if((tid1-1)>=0 && (tid2-1) >=0)
				{
                                        if(mag<Nshared1[threadIdx.x+(apron/2)-1][threadIdx.y+(apron/2)-1])
                                	{ 
				              d_suppressed[tid1*width+tid2]=(float)0;
					}
				}
                                if((tid1+1)<height && (tid2+1) <width)
				{
                                        if(mag<Nshared1[threadIdx.x+(apron/2)+1][threadIdx.y+(apron/2)+1])
					{
                                                d_suppressed[tid1*width+tid2]=(float)0;
					}
				}
                        }

                        if(theta>(67.5) && theta <= 112.5) //top and bottom. 
                        {
                                if((tid1-1)>=0)
				{
                                        if(mag<Nshared1[threadIdx.x+(apron/2)-1][threadIdx.y+(apron/2)])
                        		{          
			              		d_suppressed[tid1*width+tid2]=(float)0;
					}
				}
                                if((tid1+1)<height)
				{
                                        if(mag<Nshared1[threadIdx.x+(apron/2)+1][threadIdx.y+(apron/2)])
                                	{ 
				               d_suppressed[tid1*width+tid2]=(float)0;
					}
				}
                        }
                        if(theta>(112.5) && theta <= 157.5) //Anti Diagonal.
                        {
                                if((tid1+1)<height && (tid2-1)>=0)
				{
                                        if(mag<Nshared1[threadIdx.x+(apron/2)+1][threadIdx.y+(apron/2)-1])
					{
                                                d_suppressed[tid1*width+tid2]=(float)0;
					}
				}
                                if((tid1-1) >=0 &&(tid2+1)<=width)
				{
                                        if(mag<Nshared1[threadIdx.x+(apron/2)-1][threadIdx.y+(apron/2)+1])
					{
                                                d_suppressed[tid1*width+tid2]=(float)0;
					}
				}


                        }
        }
	
}

__global__
void gpu_verticalEdgeKeeping(float *d_hyst,float *d_I_angle,float *d_image,int h_im_width,int h_im_height,int h_alpha,float *d_HRV,float *d_map){

int tid = threadIdx.y+blockIdx.y*blockDim.y;
int vertical,diagonal,anti_diagonal;
int vert_start=0;
float theta=90;
int direction=0;
int count=0;
int ND=0;
int k =0,m=0,n;
int start_x =0,start_x_amp=0,start_y_amp=0;
int end_x =0,end_x_amp=0,end_y_amp=0;
int rodsize=0;

//Enclose everything inside if(tid <h_im_width)

if(tid <h_im_width) {
	while(vert_start<h_im_height) //not to exceed the height of the image
	{
		while(d_hyst[vert_start*h_im_width + tid]==0 && vert_start<h_im_height) //Hunting for the start of the candidate rod
		{
			vert_start++;
		}
		start_x=vert_start; //Found the head of the vertical rod
	
		while(d_hyst[vert_start*h_im_width + tid]!=0 && vert_start < h_im_height) //Traverse that candidate rod and find its end
		{
			//to get the start point of the rod
			end_x=vert_start;
			vert_start++;
		}

		//Rod traversed. Head of the vertical rod is start_x and tail is end_x
	
		rodsize = end_x - start_x + 1;	
	
		//Check the orientation
	
		vertical=0;diagonal=0,anti_diagonal=0;
	
		for(k=start_x;k<=end_x;k++)
		{
			//Fill the edge map. Initially each candidate rod IS a rod
				
			d_map[k*h_im_width + tid]=(float)255;

			theta=d_I_angle[k*h_im_width + tid];
				
			theta= (180/PI)*theta;
		
			if(theta<0)
				theta+=(float)180;

			if(theta > 22.5 && theta <=67.5) //diagonal +1
				diagonal++;
			else if(theta>112.5 && theta<=157.5) //anti-diagonal -1
				anti_diagonal++;
			else if(theta>67.5 && theta<=112.5) //vertical 0
				vertical++;

		} 

		direction= gpu_compare(vertical,diagonal,anti_diagonal);
		
		//Orientation check finished
	
		ND=0;
	
		//Qualify or disqualify the candidate rod

		if(vertical==0 && diagonal==0 && anti_diagonal==0)
		{
			//Unkown direction for Vertical rod. Disqualifying the candidate rod by setting ND=1
			ND=1;
		}
	
		//If the candidate rod is disqualified, note in the map using weak pixel (125)
		if(ND==1)
		{
			for(k=start_x;k<=end_x;k++)
			{
				d_map[k*h_im_width+tid]=(float)125;
			}
		}	
	
		//Replicate the qualified rods		
		if(direction==0 && ND==0) //replicate vertically
		{
			//These are for amplified image
			start_x_amp=h_alpha*start_x; 
			start_y_amp=h_alpha*tid;
				
			end_x_amp=h_alpha*end_x;
			end_y_amp=h_alpha*tid;
				
			count=0;

			for(k=0;k<h_alpha;k++) //no. of times (alpha) to replicate
			{		
				for(m=0;m<rodsize;m++)
				{			
					if((start_x_amp+k*rodsize+m)<h_alpha*h_im_height && start_y_amp<h_alpha*h_im_width)
						d_HRV[((start_x_amp+k*rodsize+m)*h_alpha*h_im_width) + start_y_amp]=(float)d_image[(start_x+m)*h_im_width + (tid)];
					
					//Bottom pixels need to be filled
					for(n=1;n<=(h_alpha-1);n++)
						if((start_x_amp+k*rodsize+m)<h_alpha*h_im_height && (start_y_amp+n)<h_alpha*h_im_width && (tid+1)<h_im_width)
							d_HRV[(start_x_amp+k*rodsize+m)*h_alpha*h_im_width + start_y_amp+n]=(float)d_image[(start_x+(int)(count/h_alpha))*h_im_width + (tid+1)];
					count++;		
				}
			}
		}
		
		else if(direction==1 && ND==0) //diagonal
		{

			start_x_amp=h_alpha*start_x; 
			start_y_amp=h_alpha*tid + direction*(h_alpha -1);
			end_x_amp=h_alpha*end_x;
			end_y_amp=h_alpha*tid;

			count=0;
			for(k=0;k<h_alpha;k++) //Number of times to repeat this rod
			{
				for(m=0;m<rodsize;m++)
				{
					if((start_x_amp + k*rodsize+m)<h_alpha*h_im_height &&(start_y_amp-1*k*direction)<h_alpha*h_im_width&&(start_y_amp-1*k*direction)>=0)
					{
			 			d_HRV[(start_x_amp + k*rodsize+m)*h_alpha*h_im_width + (start_y_amp-1*k*direction)]=(float)d_image[(start_x+m)*h_im_width + tid];
					}

					//if k=0, fill pixels to the left
					if(k==0)
					{
						for(n=1;n<=(h_alpha-1);n++)
						{                                        
                					if((start_x_amp + k*rodsize+m)<h_alpha*h_im_height &&(start_y_amp-1*k*direction-n)>=0 && (start_y_amp-1*k*direction-n)<h_alpha*h_im_width && ((tid)-1)>=0) //copy adjacent pixel 

                                                               d_HRV[(start_x_amp + k*rodsize+m)*h_alpha*h_im_width + start_y_amp-1*k*direction-n]=(float)d_image[(start_x+(int)(count/h_alpha))*h_im_width + (tid -1)];
							 
                                                	count++;
						}	
					}

					//if k=1, fill pixels to the right
					else
					{
						for(n=1;n<=(h_alpha-1);n++)
						{
                                                        if((start_x_amp + k*rodsize+m)<h_alpha*h_im_height &&(start_y_amp-1*k*direction+n)<h_alpha*h_im_width && (start_y_amp-1*k*direction+n)>=0 && (tid+1)<h_im_width) //copy adjacent pixel
                                                                	d_HRV[(start_x_amp + k*rodsize+m)*h_alpha*h_im_width + start_y_amp-1*k*direction+n]=(float)d_image[(start_x+(int)(count/h_alpha))*h_im_width + (tid+1)];
                                                	count++;
						}
					}
				}
			}
		}//else if ends here

		else if(direction==-1 && ND==0)//anti-diagonal
		{

			start_x_amp=h_alpha*start_x; 
			start_y_amp=h_alpha*tid; //the rod direction is perpendicular to the gradient. Starts at same y position as vertical
			end_x_amp=h_alpha*end_x;
			end_y_amp=h_alpha*tid;

			count=0;
			for(k=0;k<h_alpha;k++) //Number of times to repeat this rod
			{
				for(m=0;m<rodsize;m++)
				{	
					if((start_x_amp + k*rodsize+m)<h_alpha*h_im_height &&(start_y_amp-1*k*direction)<h_alpha*h_im_width&&(start_y_amp-1*k*direction)>=0)
						d_HRV[(start_x_amp + k*rodsize+m)*h_alpha*h_im_width + (start_y_amp-1*k*direction)]=(float)d_image[(start_x+m)*h_im_width + (tid)];

					if(k==0) //fill to the right
					{
						for(n=1;n<=(h_alpha-1);n++)
                                                       	if((start_x_amp + k*rodsize+m)<h_alpha*h_im_height &&(start_y_amp-1*k*direction+n)<h_alpha*h_im_width && (start_y_amp-1*k*direction+n)>=0 && (tid+1)<h_im_width) //copy adjacent pixel
                                                               	d_HRV[(start_x_amp + k*rodsize+m)*h_alpha*h_im_width + start_y_amp-1*k*direction+n]=(float)d_image[(start_x+(int)(count/h_alpha))*h_im_width + (tid+1)];
                                                       count++;
					}
					else //fill to the left
					{
						for(n=1;n<=(h_alpha-1);n++)
                                                	if((start_x_amp + k*rodsize+m)<h_alpha*h_im_height &&(start_y_amp-1*k*direction-n)>=0 && (start_y_amp-1*k*direction-n)<h_alpha*h_im_width && (tid-1)>=0) //copy adjacent pixel
                                                                       d_HRV[(start_x_amp + k*rodsize+m)*h_alpha*h_im_width + start_y_amp-1*k*direction-n]=(float)d_image[(start_x+(int)(count/h_alpha))*h_im_width + (tid-1)];
                                                  count++;
					}
				}
			}
		}
		 

	} //Outermost while loop

} //End the if statement on all threads
		
} //function ends here


__global__
void gpu_horizontalEdgeKeeping(float *d_hyst,float *d_I_angle,float *d_image,int h_im_width,int h_im_height,int h_alpha,float *d_HRH,float *d_maph){


int tid = threadIdx.x+blockIdx.x*blockDim.x;
int k,m,n;
int horizontal,diagonal,anti_diagonal;
int horizontal_start=0;
float theta=0;
int direction=0;
int start_x=0,start_y=0,end_x=0,end_y=0;
int start_x_amp=0,start_y_amp=0; //Start and stop positions for filling in the HR Image
int count=0;
float ND=0;
int rodsize=0;

if(tid <h_im_height){
	while(horizontal_start<h_im_width) //not to exceed the width of the image
		{
			//go to the start of the horizontal rod
			while(d_hyst[tid*h_im_width+horizontal_start]==0 && horizontal_start<h_im_width)
			{
				horizontal_start++;
			}
			start_y= horizontal_start;

			while(d_hyst[tid*h_im_width + horizontal_start]!=0 && horizontal_start<h_im_width)
			{
				end_y = horizontal_start;
				horizontal_start++;
			}
			
			rodsize = end_y - start_y +1;


		//Check the orientation
	
			horizontal=0;diagonal=0;anti_diagonal=0;
			
			for(k=start_y;k<=end_y;k++)
			{
				//Each rod IS a rod
				d_maph[tid*h_im_width+k]=(float)255;
				theta=d_I_angle[tid*h_im_width+k];
				theta = (180/PI)*theta;
				if(theta<0)
					theta+=(float)180;

				if(theta>22.5 && theta<=67.5) //diagonal
					diagonal++;
				else if(theta>112.5 && theta<=157.5) //anti-diagonal
					anti_diagonal++;
				else if(theta>(157.5) || theta <=22.5) //horizontal
					horizontal++;
			}
			direction= gpu_compare(horizontal,diagonal,anti_diagonal);
			
			ND = 0;

			if(horizontal==0 && diagonal==0 && anti_diagonal==0)
			{
				ND=1;
			}
			if(ND==1)
			{
				for(k=start_y;k<=end_y;k++)
				{
					d_maph[tid*h_im_width+k]=(float)125;
				}
			}
			if(direction==0 && ND==0) //replicate horizontally
			{
				start_x_amp=h_alpha*tid;
				start_y_amp=h_alpha*start_y;
				
				end_x=h_alpha*tid;
				end_y=h_alpha*end_y;
		
				count=0;

				for(k=0;k<h_alpha;k++) //no. of times to replicate
				{
					
					for(m=0;m<rodsize;m++)
					{			
						if(start_x_amp<h_alpha*h_im_height && (start_y_amp+k*rodsize+m)<h_alpha*h_im_width)
						//HR[(start_x*alpha*width) + (start_y+k*rod_y.size()+m)]=(float)255;	
							d_HRH[(start_x_amp*h_alpha*h_im_width) + (start_y_amp+k*rodsize+m)]=(float)d_image[(tid)*h_im_width + (start_y+m)];
						//Bottom pixels need to be filled
						for(n=1;n<=(h_alpha-1);n++)
							if((start_x_amp+n)<h_alpha*h_im_height && (start_y_amp+k*rodsize+m)<h_im_width && (tid+1)<h_im_height)
								d_HRH[(start_x_amp+n)*h_alpha*h_im_width + (start_y_amp+k*rodsize+m)]=(float)d_image[(tid+1)*h_im_width + (start_y+(int)(count/h_alpha))];

						count++;		
					}
				}
			}
			
			else if(direction==1 && ND==0) //diagonal direction, replicate upwards
			{

				start_x=h_alpha*tid + direction*(h_alpha-1);
				start_y=h_alpha*start_y;				

				count=0;
				for(k=0;k<h_alpha;k++)
				{
					for(m=0;m<rodsize;m++)
					{
						if((start_x_amp -1*k*direction)<h_alpha*h_im_height && (start_x_amp -1*k*direction)>=0 && (start_y_amp + k*rodsize+m)<h_alpha*h_im_width && (start_y_amp + k*rodsize+m)>=0)
							d_HRH[(start_x_amp -1*k*direction)*h_alpha*h_im_width + (start_y_amp + k*rodsize+m)]=(float)d_image[(tid)*h_im_width + start_y+m];
						//Top pixels need to be filled if k==0
						if(k==0)
							for(n=1;n<=(h_alpha-1);n++)
								if((start_x_amp-1*k*direction-n)>=0 && (start_x_amp-1*k*direction-n)<h_alpha*h_im_height && (start_y_amp+k*rodsize+m)<h_im_width && (tid-1)>=0)
									d_HRH[(start_x_amp-1*k*direction-n)*h_alpha*h_im_width + (start_y_amp+k*rodsize+m)]=(float)d_image[(tid-1)*h_im_width + (start_y+(int)(count/h_alpha))];
								
						else //fill the bottom pixels
							for(n=1;n<=(h_alpha-1);n++)        
								if((start_x_amp-1*k*direction+n)<h_alpha*h_im_height && (start_x_amp-1*k*direction+n)>=0 && (start_y_amp+k*rodsize+m)<h_im_width && (tid+1)<h_im_height)

                                                                        d_HRH[(start_x_amp-1*k*direction+n)*h_alpha*h_im_width + (start_y_amp+k*rodsize+m)]=(float)d_image[(tid+1)*h_im_width + (start_y+(int)(count/h_alpha))];	
	
						count++;
					}
				}

			

			}
		
			else if(direction==-1 &&ND==0)//anti-diagonal direction, replicate downwards 
			{
				start_x=h_alpha*tid;
				start_y=h_alpha*start_y;
				count=0;


				for(k=0;k<h_alpha;k++)
				{

					for(m=0;m<rodsize;m++)
					{
						if((start_x_amp -1*k*direction)<h_alpha*h_im_height && (start_x_amp -1*k*direction)>=0 && (start_y_amp + k*rodsize+m)<h_alpha*h_im_width && (start_y_amp + k*rodsize+m)>=0)
		
							d_HRH[(start_x_amp-1*k*direction)*h_alpha*h_im_width + (start_y_amp + k*rodsize + m)]=(float)d_image[(tid)*h_im_width + start_y];
				
						if(k==0) //fill bottom pixels
							for(n=1;n<=(h_alpha-1);n++)                                                             
								if((start_x_amp-1*k*direction+n)<h_alpha*h_im_height && (start_x_amp-1*k*direction+n)>=0 && (start_y_amp+k*rodsize+m)<h_im_width && (tid+1)<h_im_height)
                                                	        	d_HRH[(start_x_amp-1*k*direction+n)*h_alpha*h_im_width + (start_y_amp+k*rodsize+m)]=(float)d_image[(tid+1)*h_im_width + (start_y+(int)(count/h_alpha))];


						else //fille the top
							for(n=1;n<=(h_alpha-1);n++)                                                             
								if((start_x_amp-1*k*direction-n)>=0 && (start_x_amp-1*k*direction-n)<h_alpha*h_im_width && (start_y_amp+k*rodsize+m)<h_im_width && (tid-1)>=0)
                                                                        d_HRH[(start_x_amp-1*k*direction-n)*h_alpha*h_im_width + (start_y_amp+k*rodsize+m)]=(float)d_image[(tid-1)*h_im_width + (start_y+(int)(count/h_alpha))];
						count++;
					}
				}		

			}
		}
	}
}



//device shared for d_hyst
__shared__ float Nshared2[BLOCKWIDTH+2*(apron/2)][BLOCKWIDTH+2*(apron/2)];
__device__
void prepNshared2(float *image,int width,int height){

                int i,j;
                //NORTH
                if(blockIdx.x==0)
                {
                        if(threadIdx.x >= BLOCKWIDTH - apron/2)
                        {
                                Nshared2[threadIdx.x+apron/2 - BLOCKWIDTH][threadIdx.y+apron/2]=0;
                        }
                }
                else
                {
                        if(threadIdx.x >=BLOCKWIDTH - apron/2)
                        {
                                i=threadIdx.x+(blockIdx.x-1)*blockDim.x;
                                j=threadIdx.y+blockIdx.y*blockDim.y;
                                Nshared2[threadIdx.x+apron/2 - BLOCKWIDTH][threadIdx.y+apron/2]=
                                image[i*width+j];
                        }
                }

                //for south elements

                if(blockIdx.x==gridDim.x - 1)
                {
                        if(threadIdx.x < apron/2)
                        {
                                 Nshared2[threadIdx.x + BLOCKWIDTH + apron/2][threadIdx.y+apron/2]=0; //VKP:Initially theadIdx.x + BLOCKWIDTH - apron/2
                        }
                }
                else
                {
                        if(threadIdx.x < apron/2)
                        {
                                i= threadIdx.x +(blockIdx.x+1)*blockDim.x;
                                j=threadIdx.y+(blockIdx.y *blockDim.y);
                                Nshared2[threadIdx.x + BLOCKWIDTH + apron/2][threadIdx.y+apron/2]=image[i*width+j];//VKP:Initially theadIdx.x + BLOCKWIDTH - apron/2
                        }
                }
                //for west elements
if(blockIdx.y==0)
                {
                        if(threadIdx.y>= BLOCKWIDTH-apron/2)
                        {
                                Nshared2[threadIdx.x +apron/2][threadIdx.y+apron/2 - BLOCKWIDTH]=0;
                        }
                }
                else
                {
                        if(threadIdx.y>= BLOCKWIDTH-apron/2)
                        {
                                i= threadIdx.x +blockIdx.x *blockDim.x;
                                j = threadIdx.y+(blockIdx.y-1)*blockDim.y;
                                Nshared2[threadIdx.x +apron/2][threadIdx.y+apron/2 - BLOCKWIDTH]=image[i*width+j];
                        }
                }


                //for east elements

                if(blockIdx.y==gridDim.y-1)
                {
                        if(threadIdx.y < apron/2) //VKP:Initially BLOCKWIDTH-apron/2
                        {
                                Nshared2[threadIdx.x +apron/2][threadIdx.y+apron/2+BLOCKWIDTH]=0;
                        }
                }
                else
                {
                        if(threadIdx.y < apron/2) //VKP:Initially BLOCKWIDTH-apron/2
                        {
                                i=threadIdx.x+blockIdx.x*blockDim.x;
                                j = threadIdx.y +(blockIdx.y+1)*blockDim.y;
                                Nshared2[threadIdx.x +apron/2][threadIdx.y+apron/2+BLOCKWIDTH]=image[i*width+j];
                        }
                }

                //for north west elements

                if(blockIdx.x==0 || blockIdx.y==0)
                {
                        if((threadIdx.x >= BLOCKWIDTH-apron/2)&&(threadIdx.y >= BLOCKWIDTH - apron/2))
                        {
                                Nshared2[threadIdx.x+apron/2-BLOCKWIDTH][threadIdx.y+apron/2 - BLOCKWIDTH]=0;
                        }
                }
                else
                {
                        if((threadIdx.x >= BLOCKWIDTH-apron/2)&&(threadIdx.y >= BLOCKWIDTH - apron/2))
                        {
                                i= threadIdx.x+(blockIdx.x -1)*blockDim.x;
                                j = threadIdx.y +(blockIdx.y -1)*blockDim.y;
                                Nshared2[threadIdx.x+apron/2-BLOCKWIDTH][threadIdx.y+apron/2 - BLOCKWIDTH]=image[i*width+j];
                        }
                }
//for north east elements

                if((blockIdx.x==0) || (blockIdx.y == gridDim.y-1))
                {
                        if((threadIdx.x >= BLOCKWIDTH - apron/2)&&(threadIdx.y <apron/2))
                        {
                                Nshared2[threadIdx.x +apron/2-BLOCKWIDTH][threadIdx.y+apron/2+BLOCKWIDTH]=0; //VKP:mistake corrected in Nshared2[][*]
                        }
                }
                else
                {
                        if((threadIdx.x >= BLOCKWIDTH - apron/2)&&(threadIdx.y <apron/2))
                        {
                                i= threadIdx.x +(blockIdx.x -1)*blockDim.x;
                                j = threadIdx.y +(blockIdx.y +1)*blockDim.y;
                                Nshared2[threadIdx.x +apron/2 - BLOCKWIDTH][threadIdx.y+apron/2+BLOCKWIDTH]=image[i*width+j] =image[i*width+j]; //VKP: mistake corrected in Nshared2[][*]
                        }
                }

                //for south west elements

                if((blockIdx.x==gridDim.x-1)||(blockIdx.y==0))
                {
                        if((threadIdx.x<apron/2)&&(threadIdx.y >=BLOCKWIDTH- apron/2)) //VKP:there was a mistake with threadIdx.y condition. Check test.cu for mistake
                        {
                                Nshared2[threadIdx.x+apron/2+BLOCKWIDTH][threadIdx.y+apron/2 - BLOCKWIDTH] =0; //VKP: you forgot to apron/2
                        }
                }
                else
                {
                        if((threadIdx.x <apron/2)&&(threadIdx.y>= BLOCKWIDTH-apron/2))//VKP:there was a mistake with threadIdx.y condition. Check test.cu for mistake
                        {
                                i= threadIdx.x +(blockIdx.x +1)*blockDim.x;
                                j = threadIdx.y +(blockIdx.y -1)*blockDim.y;
                                Nshared2[threadIdx.x+apron/2+BLOCKWIDTH][threadIdx.y+apron/2 - BLOCKWIDTH] =image[i*width+j]; //you forgot to apron/2
                        }
                }

                //forsouth east elements

                if((blockIdx.x == gridDim.x -1) || (blockIdx.y == gridDim.y-1))
                {
                        if((threadIdx.x <apron/2)&&(threadIdx.y <apron/2))
                        {
                                Nshared2[threadIdx.x +apron/2+BLOCKWIDTH][threadIdx.y+apron/2+BLOCKWIDTH]=0;
                        }
                }
                else
                {
                        if((threadIdx.x <apron/2)&&(threadIdx.y <apron/2))
                        {
                                i=threadIdx.x +(blockIdx.x+1)*blockDim.x;
                                j = threadIdx.y+(blockIdx.y+1)*blockDim.y;
                        Nshared2[threadIdx.x +apron/2+BLOCKWIDTH][threadIdx.y+apron/2+BLOCKWIDTH]=image[i*width+j];
                        }
                }

                i= threadIdx.x+blockIdx.x*blockDim.x;
                j=threadIdx.y+blockIdx.y*blockDim.y;

                Nshared2[threadIdx.x +(apron/2)][threadIdx.y+(apron/2)]=image[i*width+j];
 		__syncthreads ();

}             


__global__
void gpu_meanKeeping(float *d_image,float *d_hyst,int h_im_width,int h_im_height,int h_alpha,float *d_map,float *d_maph,float *d_high_res){

float patch3_00=0;
float patch3_01=0;
float patch3_02=0;
float patch3_10=0;
float patch3_11=0;
float patch3_12=0;
float patch3_20=0;
float patch3_21=0;
float patch3_22=0;
float patcha_00=0;
float patcha_01=0;
float patcha_10=0;
float patcha_11=0;

float sum=0;
int start_x=0;
int start_y=0;
int m=0,n=0;
int tid1 = threadIdx.x+blockIdx.x*blockDim.x;
int tid2= threadIdx.y+blockIdx.y*blockDim.y;

if (tid1 < h_im_height && tid2 < h_im_width)
{
	
	prepNshared1(d_image,h_im_width,h_im_height);
	prepNshared2(d_hyst,h_im_width,h_im_height);

	start_x = h_alpha*tid1;
	start_y = h_alpha*tid2;
	
	if(d_hyst[tid1*h_im_width +tid2] != (float)255 || d_map[tid1 * h_im_width+tid2]==(float)125 || d_maph[tid1*h_im_width+tid2]==(float)125)
	{
				patch3_00 = Nshared1[(threadIdx.x+1-1)][threadIdx.y+1-1];
				patch3_01 = Nshared1[(threadIdx.x+1-1)][threadIdx.y+1];
				patch3_02 = Nshared1[(threadIdx.x+1-1)][threadIdx.y+2];
				patch3_10 = Nshared1[(threadIdx.x+1)][threadIdx.y+1-1];
				patch3_11 = Nshared1[(threadIdx.x+1)][threadIdx.y+1];
				patch3_12 = Nshared1[(threadIdx.x+1-1)][threadIdx.y+2];
				patch3_20 = Nshared1[(threadIdx.x+2)][threadIdx.y+1-1];
				patch3_21 = Nshared1[(threadIdx.x+2)][threadIdx.y+1];
				patch3_22 = Nshared1[(threadIdx.x+2)][threadIdx.y+2];	
			
		if(( tid1-1)>=0 && (tid1-1)<h_im_height && (tid2 -1)<h_im_width && (tid2-1)>=0)
		{
			if(Nshared2[threadIdx.x+1-1][threadIdx.y+1-1]==255){
			patch3_00= patch3_11;
			}
		}
		
		if((tid1-1)>=0 && (tid1-1)<h_im_height && (tid2 >= 0) &&( tid2 < h_im_width))
		{
			if(Nshared2[threadIdx.x+1-1][threadIdx.y+1]==255){
			patch3_01= patch3_11;
			}
		}

		if((tid1-1)>=0 &&(tid1-1)<h_im_height && (tid2+1)>=0 && (tid2+1)<h_im_width)
		{
			if(Nshared2[threadIdx.x+1-1][threadIdx.y+2]==255){
			patch3_02=patch3_11;
			}
		}

		if((tid1 >=0)&& (tid1)<h_im_height && (tid2-1)>=0 && (tid2-1) <h_im_width)
		{
			if(Nshared2[threadIdx.x+1][threadIdx.y+1-1]==255){
			patch3_10=patch3_11;
			}
		}

		if((tid1 >= 0)&& (tid1 <h_im_height) && (tid2 >=0) && (tid2 < h_im_width))
		{
			if(Nshared2[threadIdx.x+1][threadIdx.y+1]==255){
			patch3_11=patch3_11;
			}
		}

		if((tid1 >=0) && (tid1 < h_im_height) && (tid2+1)>=0 && (tid2+1) <h_im_width)
		{
			if(Nshared2[threadIdx.x+1][threadIdx.y+2]==255){
			patch3_12=patch3_11;
			}
		}

		if((tid1+1)>=0 && (tid1+1)<h_im_width && (tid2-1)>=0 && (tid2 -1)<h_im_width)
		{
			if(Nshared2[threadIdx.x+2][threadIdx.y+1-1]==255){
			patch3_20= patch3_11;
			}
		}

		if((tid1+1)>=0 && (tid1+1)<h_im_height && (tid2>=0) && (tid2<h_im_width))
		{
			if(Nshared2[threadIdx.x+2][threadIdx.y+1]==255){
			patch3_21= patch3_11;
			}
		}

		if((tid1+1)>=0 && (tid1+1)<h_im_height && (tid2+1)>=0 && (tid2+1)<h_im_width)
		{
			if(Nshared2[threadIdx.x+2][threadIdx.y+2]==255){
			patch3_22=patch3_11;
			}
		}
			
		if(( tid1-1)>=0 && (tid1-1)<h_im_height && (tid2 -1)<h_im_width && (tid2-1)>=0)
		{
			if(Nshared2[threadIdx.x+1-1][threadIdx.y+1-1]==255){
			patch3_00= (patch3_10 + patch3_01)/2;
			}
		}
	
		if((tid1-1)>=0 && (tid1-1)<h_im_height && (tid2 >= 0) &&( tid2 < h_im_width))
		{
			if(Nshared2[threadIdx.x+1-1][threadIdx.y+1]==255){
			patch3_01=(patch3_11 +patch3_01)/2;
			}
		}

		
		if((tid1-1)>=0 &&(tid1-1)<h_im_height && (tid2+1)>=0 && (tid2+1)<h_im_width)
		{
			if(Nshared2[threadIdx.x+1-1][threadIdx.y+2]==255){
			patch3_02=(patch3_12 +patch3_01)/2;
			}
		}

		if((tid1 >=0)&& (tid1)<h_im_height && (tid2-1)>=0 && (tid2-1) <h_im_width)
		{
			if(Nshared2[threadIdx.x+1][threadIdx.y+1-1]==255){
			patch3_10=(patch3_10 +patch3_11)/2;
			}
		}

		if((tid1 >= 0)&& (tid1 <h_im_height) && (tid2 >=0) && (tid2 < h_im_width))
		{
			if(Nshared2[threadIdx.x+1][threadIdx.y+1]==255){
			patch3_11= (patch3_11 + patch3_11)/2;
			}
		}

		if((tid1 >=0) && (tid1 < h_im_height) && (tid2+1)>=0 && (tid2+1) <h_im_width)
		{
			if(Nshared2[threadIdx.x+1-1][threadIdx.y+2]==255){
			patch3_12=(patch3_12 +patch3_11)/2;
			}
		}

		
		if((tid2+1)>=0 && (tid1+1)<h_im_width && (tid2-1)>=0 && (tid2 -1)<h_im_width)
		{
			if(Nshared2[threadIdx.x+2][threadIdx.y+1-1]==255){
			patch3_20=(patch3_10 +patch3_21)/2;
			}
		}

		if((tid1+1)>=0 && (tid1+1)<h_im_height && (tid2>=0) && (tid2<h_im_width))
		{
			if(Nshared2[threadIdx.x+2][threadIdx.y+1]==255){
			patch3_21=(patch3_11 +patch3_21)/2;
			}
		}


		if((tid1+1)>=0 && (tid1+1)<h_im_height && (tid2+1)>=0 && (tid2+1)<h_im_width)
		{
			if(Nshared2[threadIdx.x+2][threadIdx.y+2]==255){
			patch3_22=(patch3_12 +patch3_21)/2;
			}
		}
		
patcha_00 = (patch3_00 +patch3_01+patch3_10+patch3_11)/4;
patcha_01=(patch3_01 +patch3_02+patch3_11+patch3_12)/4;
patcha_10 =(patch3_10+patch3_11+patch3_20+patch3_21)/4;
patcha_11 =(patch3_11+patch3_12+patch3_21+patch3_22)/4;			

sum = (patcha_00 +patcha_01+patcha_10+patcha_11);

patcha_00 = (patcha_00/sum)*h_alpha*h_alpha*d_image[tid1*h_im_width + tid2];
patcha_01=(patcha_01/sum)*h_alpha*h_alpha*d_image[tid1*h_im_width +tid2];
patcha_10= (patcha_10/sum)*h_alpha*h_alpha*d_image[tid1*h_im_width +tid2];
patcha_11=(patcha_11/sum)*h_alpha*h_alpha*d_image[tid1*h_im_width +tid2];

d_high_res[(start_x)*h_alpha*h_im_width+(start_y)]=patcha_00;
d_high_res[(start_x)*h_alpha*h_im_width+(start_y+1)]=patcha_01;
d_high_res[(start_x+1)*h_alpha*h_im_width+start_y]=patcha_10;
d_high_res[(start_x+1)*h_alpha*h_im_width+(start_y+1)]=patcha_11;
	
}

	else if(d_hyst[tid1*h_im_width+tid2]==(float)255)
	{
		sum =0;
		for(m=0;m<h_alpha;m++)
		{
			for(n=0;n<h_alpha;n++)
			{
				sum+=d_high_res[(start_x+m)*h_alpha*h_im_width+(start_y+n)];
			}
		}
		for(m=0;m<h_alpha;m++)
		{
			for(n=0;n<h_alpha;n++)
			{
				d_high_res[(start_x+m)*h_alpha*h_im_width +(start_y +n)]=(d_high_res[(start_x+m)*h_alpha*h_im_width+(start_y+n)]/sum)*(h_alpha*h_alpha)*d_image[tid1*h_im_width+tid2];
			}
		}
	}
	
  }
}



//commbining hrh and hrv

__global__
void gpu_combineHRH_HRV(float *d_high_res,float *d_HRH,float *d_maph,float *d_map,int h_alpha,int h_im_height,int h_im_width)
{

int m=0,n=0;
int tid1 = threadIdx.x+blockIdx.x*blockDim.x;
int tid2= threadIdx.y+blockIdx.y*blockDim.y;

if(tid1 < h_im_height && tid2 < h_im_width)
{
	for(m=0;m<h_alpha;m++)
	{
		for(n=0;n<h_alpha;n++)
		{	
			if(((tid1*h_alpha+m) <h_alpha*h_im_height) &&((tid2*h_alpha+n)*h_alpha*h_im_width))
			{
				d_high_res[(tid1*h_alpha+m)*h_im_width*h_alpha +(tid2*h_alpha+n)]+=d_HRH[(tid1*h_alpha+m)*h_im_width*h_alpha+(tid2*h_alpha+n)];
				if((d_maph[tid1*h_im_width+tid2] == d_map[tid1*h_im_width+tid2]) && d_maph[tid1*h_im_width +tid2]==255)
				{
					d_maph[tid1*h_im_width+tid2]=125;
					d_map[tid1*h_im_width+tid2]=125;
				}
				else if((d_maph[tid1*h_im_width+tid2]==255 && d_map[tid1*h_im_width+tid2] !=255) || (d_maph[tid1*h_im_width+tid2] != 255 && d_map[tid1*h_im_width+tid2]==255))	
				{
					d_high_res[(tid1*h_alpha+m)*h_im_width*h_alpha+(tid2*h_alpha+n)] = d_high_res[(tid1*h_alpha+m)*h_im_width*h_alpha+(tid2*h_alpha+n)]+ d_HRH[(tid1*h_alpha+m)*h_im_width*h_alpha+(tid2*h_alpha+n)];
				}
			}
		}
	}
}


}




void consolidated_convolveMagAngleSuppressionSortVHM(float *vmap,float *hmap,float *h_image,int h_im_width,int h_im_height,float *h_gx_mask,float *h_gy_mask,int h_m_w,float *h_I_angle,float *h_hyst,float *h_gxy,int h_alpha,float *h_HRV,float *h_HRH,float *h_highres){

float *d_image,*d_Gxy_outimage,*d_suppressed,*d_Igx_mask,*d_Igy_mask,*d_I_angle,*d_hyst,*d_HRV,*d_map,*d_HRH,*d_maph,*d_high_res;

//Allocate memory for device pointers
hipMalloc((void **)&d_I_angle,sizeof(float)*h_im_width*h_im_height);
hipMalloc((void **)&d_Gxy_outimage,sizeof(float)*h_im_width*h_alpha*h_im_height*h_alpha);
hipMalloc((void **)&d_image,sizeof(float)*h_im_width*h_im_height);
hipMalloc((void **)&d_suppressed,sizeof(float)*h_im_width*h_im_height);
hipMalloc((void **)&d_Igx_mask,sizeof(float)*h_m_w*h_m_w);
hipMalloc((void **)&d_Igy_mask,sizeof(float)*h_m_w*h_m_w);

//memcpy to device pointers
hipMemcpy(d_image,h_image,sizeof(float)*h_im_width*h_im_height,hipMemcpyHostToDevice);
hipMemcpy(d_Igx_mask,h_gx_mask,sizeof(float)*h_m_w*h_m_w,hipMemcpyHostToDevice);
hipMemcpy(d_Igy_mask,h_gy_mask,sizeof(float)*h_m_w*h_m_w,hipMemcpyHostToDevice);

dim3 dimGrid(ceil((float)h_im_height/(float)BLOCKWIDTH),ceil((float)h_im_width/(float)BLOCKWIDTH),1);
dim3 dimBlock(16,16,1);
gpu_conv_mag_phase<<<dimGrid,dimBlock>>>(d_image,h_im_width,h_im_height,d_Gxy_outimage,d_Igx_mask,d_Igy_mask,d_I_angle,d_suppressed);

gpu_suppress<<<dimGrid,dimBlock>>>(d_Gxy_outimage,d_suppressed,d_I_angle,h_im_width,h_im_height);

hipFree(d_Gxy_outimage);
hipFree(d_Igx_mask);
hipFree(d_Igy_mask);


thrust::device_ptr<float> thr_d(d_suppressed);
thrust::device_vector<float>d_supp_vec(thr_d,thr_d+(h_im_height*h_im_width));
thrust::host_vector<float>h_supp_vec(h_im_height*h_im_width);
thrust::sort(d_supp_vec.begin(),d_supp_vec.end());

hipMalloc((void **)&d_hyst,sizeof(float)*h_im_width*h_im_height);
hipMalloc((void **)&d_HRV,sizeof(float)*(h_im_width*h_alpha)*(h_im_height*h_alpha));
hipMalloc((void **)&d_HRH,sizeof(float)*(h_im_width*h_alpha)*(h_im_height*h_alpha));
hipMalloc((void **)&d_map,sizeof(float)*h_im_width*h_im_height);
hipMalloc((void **)&d_maph,sizeof(float)*h_im_width*h_im_height);
hipMalloc((void **)&d_high_res,sizeof(float)*(h_im_width *h_alpha)*(h_im_width*h_alpha));

hipMemset(d_high_res,0,sizeof(float)*h_im_width*h_alpha*h_im_width*h_alpha);
hipMemset(d_HRV,0,sizeof(float)*(h_im_width*h_alpha)*(h_im_height*h_alpha));
hipMemset(d_HRH,0,sizeof(float)*(h_im_width*h_alpha)*(h_im_height*h_alpha));
hipMemset(d_map,0,sizeof(float)*(h_im_width)*(h_im_height));
hipMemset(d_maph,0,sizeof(float)*(h_im_width)*(h_im_height));


//getting high and low threshold

float index = float(0.90)*h_im_height*h_im_width;
float th_high = d_supp_vec[(int)index];
float th_low =th_high/5;

//Double-thresholding
gpu_doubleThreshold<<<dimGrid,dimBlock>>>(d_hyst,d_suppressed,h_im_width,h_im_height,th_high,th_low);

//Edge linking
gpu_edgeLinking<<<dimGrid,dimBlock>>>(d_suppressed,d_hyst,h_im_width,h_im_height);

//Vertical edge keeping
dim3 dimGrid1(1,ceil((float)h_im_width/(float)BLOCKWIDTH),1);
dim3 dimBlock1(1,256,1);
gpu_verticalEdgeKeeping<<<dimGrid1,dimBlock1>>>(d_hyst,d_I_angle,d_image,h_im_width,h_im_height,h_alpha,d_HRV,d_map);


//Horizontal edge keeping
dim3 dimGrid2(ceil((float)h_im_width/(float)BLOCKWIDTH),1,1);
dim3 dimBlock2(512,1,1);
gpu_horizontalEdgeKeeping<<<dimGrid2,dimBlock2>>>(d_hyst,d_I_angle,d_image,h_im_width,h_im_height,h_alpha,d_HRH,d_maph);


d_high_res=d_HRV;
gpu_combineHRH_HRV<<<dimGrid,dimBlock>>>(d_high_res,d_HRH,d_maph,d_map,h_alpha,h_im_height,h_im_width);

gpu_meanKeeping<<<dimGrid,dimBlock>>>(d_image,d_hyst,h_im_width,h_im_height,h_alpha,d_map,d_maph,d_high_res);

hipMemcpy(h_highres,d_high_res,sizeof(float)*(h_im_width *h_alpha)*(h_im_height * h_alpha),hipMemcpyDeviceToHost);


hipFree(d_image);
hipFree(d_I_angle);
hipFree(d_suppressed);
hipFree(d_hyst);
hipFree(d_HRV);
hipFree(d_map);
hipFree(d_HRH);
hipFree(d_maph);

}

